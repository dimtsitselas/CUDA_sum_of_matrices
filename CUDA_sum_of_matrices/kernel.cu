
#include "hip/hip_runtime.h"


#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void addMatricesCuda(int* a, int* b, int* c, int nRows, int nCols)
{
    int baseI = (blockDim.x * blockIdx.x + threadIdx.x) * nCols;

    for (int i = baseI; i < baseI + nCols; i++)
    {
        if (i < nRows * nCols)
            c[i] = a[i] + b[i];
    }
}

void addMatrices(int* a, int* b, int* c, int nRows, int nCols)
{
    // create device pointers
    int* d_a, * d_b, * d_c; 
    
    int nBytes = nRows * nCols * sizeof(int);

    hipMalloc(&d_a, nBytes);
    hipMalloc(&d_b, nBytes);
    hipMalloc(&d_c, nBytes);

    hipMemcpy(d_a, a, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, nBytes, hipMemcpyHostToDevice);

    addMatricesCuda <<<ceil(nRows / 256.0), 256.0 >>> (d_a, d_b, d_c, nRows, nCols);

    hipMemcpy(c, d_c, nBytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void addMatricesHost(int* a, int* b, int* c, int nRows, int nCols)
{
    for (int i = 0; i < nRows * nCols; i++)
        c[i] = a[i] + b[i];
}

int main()
{
    int* a, * b, * c_h, * c_d, nCols, nRows;
    nCols = nRows = 1000; 

    a = (int*)malloc(nCols * nRows * sizeof(int));
    b = (int*)malloc(nCols * nRows * sizeof(int));
    c_h = (int*)malloc(nCols * nRows * sizeof(int));
    c_d = (int*)malloc(nCols * nRows * sizeof(int));

    for (int i = 0; i < nRows; i++)
    {
        for (int j = 0; j < nCols; j++)
        {
            if(a) a[i * nCols + j] = rand() % 100;
            if(b) b[i * nCols + j] = rand() % 100;
        }
    }

    addMatrices(a, b, c_d, nRows, nCols);
    addMatricesHost(a, b, c_h, nRows, nCols);

    for (int i = 0; i < nRows * nCols; i++)
        assert(c_d[i] == c_h[i]);

    printf("OK\n");

    return 0;
}
